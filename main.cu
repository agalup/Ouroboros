#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>

#include "device/Ouroboros_impl.cuh"
#include "device/MemoryInitialization.cuh"
#include "InstanceDefinitions.cuh"
#include "Utility.h"
#include "hip/hip_runtime.h"
#include "pmm-utils.cuh"

using namespace std;

//#define DEBUG
#ifdef DEBUG
#define debug(a...) printf(a)
#else
#define debug(a...)
#endif


//producer
template <typename MemoryManagerType>
__global__
void mem_manager(volatile int* exit_signal, 
                volatile int* requests_number, 
                volatile int* request_iter,
                volatile int* request_signal, 
                volatile int* request_ids, 
                MemoryManagerType* mm,
                volatile int** d_memory,
                volatile int* request_mem_size,
                volatile int* lock){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (! exit_signal[0]){
        for (int request_id=thid; request_id<requests_number[0]; 
                request_id += blockDim.x*gridDim.x){

            if (request_signal[request_id] == 1){

                // SEMAPHORE
                acquire_semaphore((int*)lock, request_id);
                printf("mm: request recieved %d\n", request_id); 
                int req_id = atomicAdd((int*)&request_iter[0], 1);
                request_ids[request_id] = req_id;

                //__threadfence();
                //d_memory[req_id] = reinterpret_cast<volatile int*>
                //        (mm->malloc(request_mem_size[request_id]));
                //__threadfence();
                //assert(d_memory[req_id]);

                // SIGNAL update
                atomicExch((int*)&request_signal[request_id], 2);

                //__threadfence();
                release_semaphore((int*)lock, request_id);
                // SEMAPHORE

                printf("mm: request done %d\n", request_id, req_id);
                //break;
            }
        }
    }
}

//consumer
__global__
void app(volatile int* exit_signal,
         volatile int** d_memory, 
         volatile int* request_signal, 
         volatile int* request_mem_size,
         volatile int* request_id, 
         volatile int* exit_counter, 
         volatile int* lock){
    int thid = blockDim.x * blockIdx.x + threadIdx.x;

    // SEMAPHORE
    acquire_semaphore((int*)lock, thid);
    request_mem_size[thid] = 4;
    request_id[thid] = -1;
    int req_id = -1;
    atomicExch((int*)&request_signal[thid], 1);
    __threadfence();
    release_semaphore((int*)lock, thid);
    // SEMAPHORE
    
    // wait for success
    while (!exit_signal[0]){
        __threadfence();
        if (request_signal[thid] == 2){
            
            // SEMAPHORE
            acquire_semaphore((int*)lock, thid);
            req_id = request_id[thid];
            if (req_id >= 0) {
                //assert(d_memory[req_id]);
                //d_memory[req_id][0] = thid;
            }
            request_signal[thid] = 0;
            __threadfence();
            //printf("app: request %d success\n", thid);
            release_semaphore((int*)lock, thid);
            // SEMAPHORE
        
            //printf("done request(%d) = %d, request_id = %d\n", thid, req_id, request_id[thid]);
            break;
        }
    }
    atomicAdd((int*)&exit_counter[0], 1);
    
}

int main(int argc, char *argv[]){

    //Ouroboros initialization
    size_t instantitation_size = 7168ULL * 1024ULL * 1024ULL;
    using MemoryMangerType = OuroVACQ;
    MemoryMangerType memory_manager;
    memory_manager.initialize(instantitation_size);

//Creat two asynchronous streams which may run concurrently with the default stream 0.
    //The streams are not synchronized with the default stream.
    hipStream_t mm_stream, app_stream;
    //hipError_t retval;
    GUARD_CU(hipStreamCreateWithFlags( &mm_stream, hipStreamNonBlocking));
    GUARD_CU(hipStreamCreateWithFlags(&app_stream, hipStreamNonBlocking));
    
    int* exit_signal;
    GUARD_CU(hipMallocManaged(&exit_signal, sizeof(int32_t)));
    *exit_signal = 0;

    int* exit_counter;
    GUARD_CU(hipMallocManaged(&exit_counter, sizeof(uint32_t)));
    *exit_counter = 0;

    //int grid_size = 1;
    //int block_size = 32;

    int grid_size = 1;
    int block_size = 64;

    if (argc > 1){
        printf("args: %s %s\n", argv[1], argv[2]);
        grid_size = atoi(argv[1]);
        block_size = atoi(argv[2]);
    }
   /* 
    int man_grid_size, man_block_size;
    GUARD_CU(hipOccupancyMaxPotentialBlockSize(&man_grid_size, &man_block_size, mem_manager<MemoryMangerType>));
    printf("Manager: Max Potential Occupancy: G:%d, B:%d, gives %d threads\n", \
    man_grid_size, man_block_size, man_grid_size*man_block_size); 
     
    int app_grid_size, app_block_size;
    GUARD_CU(hipOccupancyMaxPotentialBlockSize(&app_grid_size, &app_block_size, app));
    printf("APP: Max Potential Occupancy: G:%d, B:%d, gives %d threads\n", \
    app_grid_size, app_block_size, app_grid_size*app_block_size); 
  
    int grid_size = min(man_grid_size, app_grid_size);
    int block_size = min(man_block_size, app_block_size);
*/
    int requests_num{grid_size*block_size};
    std::cout << "Number of Allocations: " << requests_num << "\n";

    volatile int** d_memory{nullptr};
    GUARD_CU(hipMalloc(&d_memory, sizeof(volatile int*) * requests_num));

    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());

    //Request auxiliary
    RequestType requests;
    requests.init(requests_num);
    requests.memset();
    GUARD_CU(hipPeekAtLastError());

    //Run presistent kernel (Memory Manager)
    mem_manager<<<grid_size, block_size, 0, mm_stream>>>(exit_signal,
    requests.requests_number, 
    requests.request_iter, 
    requests.request_signal, 
    requests.request_id,
    memory_manager.getDeviceMemoryManager(),
    d_memory,
    requests.request_mem_size,
    /*requests.request_success,*/
    requests.lock);

    //GUARD_CU(hipStreamSynchronize( mm_stream));
    GUARD_CU(hipPeekAtLastError());

    int isRunning = 0;
    int old_counter = 0;
    while (1){
        if (exit_counter[0] == block_size*grid_size){
            //printf("break because of exit_counter = %d\n", exit_counter[0]);
            *exit_signal = 1;
            /*GUARD_CU(hipStreamSynchronize(app_stream));
            GUARD_CU(hipStreamSynchronize(mm_stream));
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());
            test1<<<grid_size, block_size, 0, app_stream>>>(d_memory);
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());
            mem_test((int**)d_memory, requests_num, grid_size, block_size, mm_stream);*/
            break;
        }else{
            if (exit_counter[0] != old_counter){
                old_counter = exit_counter[0];
                printf("no break, exit_counter = %d\n", exit_counter[0]);
            }
        }
        if (!isRunning){
            //Run application
            app<<<grid_size, block_size, 0, app_stream>>>(exit_signal, d_memory, 
            requests.request_signal, 
            requests.request_mem_size, 
            requests.request_id, 
            exit_counter, 
            requests.lock);

            GUARD_CU(hipPeekAtLastError());
            isRunning = 1;
        }
    }

    GUARD_CU(hipStreamSynchronize(mm_stream));
    GUARD_CU(hipStreamSynchronize(app_stream));
    printf("DONE!\n");
    return 0;
}

