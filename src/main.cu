#include "hip/hip_runtime.h"
#include <iostream>

#include "device/Ouroboros_impl.cuh"
#include "device/MemoryInitialization.cuh"
#include "InstanceDefinitions.cuh"
#include "Utility.h"

#define TEST_MULTI

template <typename MemoryManagerType>
__global__ void d_testAllocation(MemoryManagerType* mm, int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;

	verification_ptr[tid] = reinterpret_cast<int*>(mm->malloc(allocation_size));
}


// run 1 thread per warp: group allocation for entire warp
template <typename MemoryManagerType>
__global__ void d_test_warp_Allocation(MemoryManagerType* mm, int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;

    if ((threadIdx.x % 32) == 0)
	    verification_ptr[tid] = reinterpret_cast<int*>(mm->malloc(32 * allocation_size));

    __syncthreads();
    verification_ptr[tid] = reinterpret_cast<int*>(reinterpret_cast<char*>(verification_ptr[(threadIdx.x/32)*32]) +
    ((threadIdx.x%32) * allocation_size));
}

__global__ void d_testWriteToMemory(int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		ptr[i] = tid;
	}
}

__global__ void d_testReadFromMemory(int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	if(threadIdx.x == 0 && blockIdx.x == 0)
		printf("Test Read!\n");
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		if(ptr[i] != tid)
		{
			printf("%d - %d | We got a wrong value here! %d vs %d\n", threadIdx.x, blockIdx.x, ptr[i], tid);
			return;
		}
	}
}

template <typename MemoryManagerType>
__global__ void d_testFree(MemoryManagerType* mm, int** verification_ptr, int num_allocations)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;

	mm->free(verification_ptr[tid]);
}


// run 1 thread per warp: group allocation for entire warp
template <typename MemoryManagerType>
__global__ void d_test_warp_Free(MemoryManagerType* mm, int** verification_ptr, int num_allocations)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;

    if (threadIdx.x % 32 == 0)
	    mm->free(verification_ptr[tid]);
}

int main(int argc, char* argv[])
{
	std::cout << "Usage: num_allocations allocation_size_in_bytes blockSize\n";
	int num_allocations {10000};
	int allocation_size_byte {16};
	int num_iterations {10};
	int blockSize {256};
    int allthreads {1};
	if(argc >= 2){
		num_allocations = atoi(argv[1]);
		if(argc >= 3){
			allocation_size_byte = atoi(argv[2]);
            if (argc >= 4){
                blockSize = atoi(argv[3]);
                if (argc >= 5){
                    allthreads = atoi(argv[4]);
                }
            }
		}
	}
	allocation_size_byte = Ouro::alignment(allocation_size_byte, sizeof(int));
	std::cout << "Number of Allocations: " << num_allocations << " | Allocation Size: " << allocation_size_byte << " | Iterations: " << num_iterations << std::endl;
    if (allthreads){
        printf("All threads per warp\n");
    }else{
        printf("One thread per warp\n");
    }

	#ifdef TEST_PAGES

	#ifdef TEST_VIRTUALARRAY
	std::cout << "Testing page-based memory manager - Virtualized Array!\n";
	#ifndef TEST_MULTI
	using MemoryManagerType = OuroVAPQ;
	#else
	using MemoryManagerType = MultiOuroVAPQ;
	#endif
	#elif TEST_VIRTUALLIST
	std::cout << "Testing page-based memory manager - Virtualized List!\n";
	#ifndef TEST_MULTI
	using MemoryManagerType = OuroVLPQ;
	#else
	using MemoryManagerType = MultiOuroVLPQ;
	#endif
	#else
	std::cout << "Testing page-based memory manager - Standard!\n";
	#ifndef TEST_MULTI
	using MemoryManagerType = OuroPQ;
	#else
	using MemoryManagerType = MultiOuroPQ;
	#endif
	#endif

	#elif TEST_CHUNKS

	#ifdef TEST_VIRTUALARRAY
	std::cout << "Testing chunk-based memory manager - Virtualized Array!\n";
	#ifndef TEST_MULTI
	using MemoryManagerType = OuroVACQ;
	#else
	using MemoryManagerType = MultiOuroVACQ;
	#endif
	#elif TEST_VIRTUALLIST
	std::cout << "Testing chunk-based memory manager - Virtualized List!\n";
	#ifndef TEST_MULTI
	using MemoryManagerType = OuroVLCQ;
	#else
	using MemoryManagerType = MultiOuroVLCQ;
	#endif
	#else
	std::cout << "Testing chunk-based memory manager - Standard!\n";
	#ifndef TEST_MULTI
	using MemoryManagerType = OuroCQ;
	#else
	using MemoryManagerType = MultiOuroCQ;
	#endif
	#endif

	#endif

	size_t instantitation_size = 4 * 1024ULL * 1024ULL * 1024ULL;
	MemoryManagerType memory_manager;
	memory_manager.initialize(instantitation_size);

	int** d_memory{nullptr};
	HANDLE_ERROR(hipMalloc(&d_memory, sizeof(int*) * num_allocations));
    printf("num_allocations %d\n", num_allocations * sizeof(int*));

	int gridSize {Ouro::divup(num_allocations, blockSize)};
	float timing_allocation{0.0f};
	float timing_free{0.0f};
	hipEvent_t start, end;
	for(auto i = 0; i < num_iterations; ++i)
	{
		start_clock(start, end);

        if (allthreads){
		    d_testAllocation <MemoryManagerType> <<<gridSize, blockSize>>>(memory_manager.getDeviceMemoryManager(), d_memory, num_allocations, allocation_size_byte);
        }else{
		    d_test_warp_Allocation <MemoryManagerType> <<<gridSize, blockSize>>>(memory_manager.getDeviceMemoryManager(), d_memory, num_allocations, allocation_size_byte);
        }

		timing_allocation += end_clock(start, end);

		HANDLE_ERROR(hipDeviceSynchronize());

		d_testWriteToMemory<<<gridSize, blockSize>>>(d_memory, num_allocations, allocation_size_byte);

		HANDLE_ERROR(hipDeviceSynchronize());

		d_testReadFromMemory<<<gridSize, blockSize>>>(d_memory, num_allocations, allocation_size_byte);

		HANDLE_ERROR(hipDeviceSynchronize());

		start_clock(start, end);

        if (allthreads){
		    d_testFree <MemoryManagerType> <<<gridSize, blockSize>>>(memory_manager.getDeviceMemoryManager(), d_memory,
            num_allocations);
        }else{
		    d_test_warp_Free <MemoryManagerType> <<<gridSize, blockSize>>>(memory_manager.getDeviceMemoryManager(), d_memory, num_allocations);
        }

		timing_free += end_clock(start, end);

		HANDLE_ERROR(hipDeviceSynchronize());
	}
	timing_allocation /= num_iterations;
	timing_free /= num_iterations;

	std::cout << "Timing Allocation: " << timing_allocation << "ms" << std::endl;
    int num_alloc_per_sec = (1000.0 * num_allocations)/timing_allocation;
    if (num_alloc_per_sec/(1000*1000*1000) > 0){
        std::cout << "# allocations per sec: " << num_alloc_per_sec/1000000000 << "G" << std::endl;
    }else if (num_alloc_per_sec/(1000*1000) > 0){
        std::cout << "# allocations per sec: " << num_alloc_per_sec/1000000 << "M" << std::endl;
    }else if (num_alloc_per_sec/1000 > 0){
        std::cout << "# allocations per sec: " << num_alloc_per_sec/1000 << std::endl;
    }else{
        std::cout << "# allocations per sec: " << num_alloc_per_sec << std::endl;
    }
	std::cout << "Timing       Free: " << timing_free << "ms" << std::endl;
	std::cout << "Testcase DONE!\n";

    return 0;
}
